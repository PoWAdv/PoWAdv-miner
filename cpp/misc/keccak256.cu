#include "hip/hip_runtime.h"
/*
 * Keccak 256
 *https://raw.githubusercontent.com/nicehash/ccminer-sp/master/keccak/keccak256.cu
 */

extern "C"
{
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_keccak.h"

//#include "miner.h"
}

#include "cuda_helper.h"

static uint32_t h_nounce[MAX_GPUS][2];

extern void keccak256_cpu_init(int thr_id, uint32_t threads);
extern void keccak256_setBlock_80(void *pdata,const uint64_t *ptarget);
extern void keccak256_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *h_nounce);

// CPU Hash
extern "C" void keccak256_hash(void *state, const void *input)
{
	sph_keccak_context ctx_keccak;

	uint32_t hash[16];

	sph_keccak256_init(&ctx_keccak);
	sph_keccak256 (&ctx_keccak, input, 80);
	sph_keccak256_close(&ctx_keccak, (void*) hash);

	memcpy(state, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_keccak256(int thr_id, uint32_t *pdata,
	const uint32_t *ptarget, uint32_t max_nonce,
	unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	uint32_t intensity = (device_sm[device_map[thr_id]] > 500) ? 1 << 28 : 1 << 27;;
	uint32_t throughput = device_intensity(device_map[thr_id], __func__, intensity); // 256*4096
	throughput = min(throughput, max_nonce - first_nonce);


	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x01;

	if (!init[thr_id]) {
		hipSetDevice(device_map[thr_id]);
		if (!opt_cpumining) hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		if (opt_n_gputhreads == 1)
		{
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		}
		keccak256_cpu_init(thr_id, (int)throughput);
		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++) {
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);
	}

	keccak256_setBlock_80((void*)endiandata, (uint64_t *)ptarget);

	do {

		keccak256_cpu_hash_80(thr_id, (int) throughput, pdata[19], h_nounce[thr_id]);
		if (h_nounce[thr_id][0] != UINT32_MAX)
		{
			uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], h_nounce[thr_id][0]);
			keccak256_hash(vhash64, endiandata);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				// check if there was some other ones...
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (h_nounce[thr_id][1] != 0xffffffff)
				{
					pdata[21] = h_nounce[thr_id][1];
					res++;
					if (opt_benchmark)
						applog(LOG_INFO, "GPU #%d Found second nounce %08x", thr_id, h_nounce[thr_id][1], vhash64[7], Htarg);
				}
				pdata[19] = h_nounce[thr_id][0];
				if (opt_benchmark)
					applog(LOG_INFO, "GPU #%d Found nounce %08x", thr_id, h_nounce[thr_id][0], vhash64[7], Htarg);
				return res;
			}
			else
			{
				if (vhash64[7] != Htarg)
				{
					applog(LOG_INFO, "GPU #%d: result for %08x does not validate on CPU!", thr_id, h_nounce[thr_id][0]);
				}
			}
		}

		pdata[19] += throughput;
	} while (!scan_abort_flag && !work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));
	*hashes_done = pdata[19] - first_nonce;
	return 0;
}
